
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Device Functions

__global__ void GPU_Vector_Times_Constant(float *a, float C, int N) {
    for (int i = 0; i < N; i++) {
        a[i] = C*a[i];
    }
}

// Host Functions

void Allocate_Memory(float **h_a, float **h_b, float **d_a, int N) {
    size_t size = N*sizeof(float);
    hipError_t Error;
    // Host memory
    *h_a = (float*)malloc(size); 
    *h_b = (float*)malloc(size); 
    // Device memory
    Error = hipMalloc((void**)d_a, size); 
    printf("CUDA error (malloc d_a) = %s\n", hipGetErrorString(Error));
}

void Free_Memory(float **h_a, float **h_b, float **d_a) {
    if (*h_a) free(*h_a);
    if (*h_b) free(*h_b);
    if (*d_a) hipFree(*d_a);
}

void Send_To_Device(float **h_a, float **d_a, int N) {
    // Size of data to send
    size_t size = N*sizeof(float);
    // Grab a error type
    hipError_t Error;

    // Send A to the GPU
    Error = hipMemcpy(*d_a, *h_a, size, hipMemcpyHostToDevice); 
    printf("CUDA error (memcpy h_a -> d_a) = %s\n", hipGetErrorString(Error));
}

void Get_From_Device(float **d_a, float **h_b, int N) {
    // Size of data to send
    size_t size = N*sizeof(float);
    // Grab a error type
    hipError_t Error;
    // Send d_a to the host variable h_b
    Error = hipMemcpy(*h_b, *d_a, size, hipMemcpyDeviceToHost);
    printf("CUDA error (memcpy d_a -> h_b) = %s\n", hipGetErrorString(Error));
}

void Vector_Times_Constant(float **d_a, float C, int N) {
    // This code is written to use a single thread on the GPU.
    // This is horrible and very inefficient.
    int threadsPerBlock = 1;
    int blocksPerGrid = 1;
    GPU_Vector_Times_Constant<<<blocksPerGrid, threadsPerBlock>>>(*d_a, C, N);
}
